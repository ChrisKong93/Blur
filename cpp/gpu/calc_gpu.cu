#include "hip/hip_runtime.h"
#include "calc_gpu.cuh"
#define Row  1024
#define Col 1024
using namespace std;

__global__ void matrix_mul_gpu(int *M, int* N, int* P, int width)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;

	int sum = 0;
	for (int k = 0; k < width; k++)
	{
		int a = M[j*width + k];
		int b = N[k*width + i];
		sum += a * b;
	}
	P[j*width + i] = sum;
}



int main()
{
	clock_t startTime, endTime;
	startTime = clock();//��ʱ��ʼ

	int *A = (int *)malloc(sizeof(int) * Row * Col);
	int *B = (int *)malloc(sizeof(int) * Row * Col);
	int *C = (int *)malloc(sizeof(int) * Row * Col);
	//malloc device memory
	int *d_dataA, *d_dataB, *d_dataC;
	hipMalloc((void**)&d_dataA, sizeof(int) * Row * Col);
	hipMalloc((void**)&d_dataB, sizeof(int) * Row * Col);
	hipMalloc((void**)&d_dataC, sizeof(int) * Row * Col);
	//set value
	for (int i = 0; i < Row * Col; i++) {
		A[i] = 90;
		B[i] = 10;
	}

	hipMemcpy(d_dataA, A, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
	hipMemcpy(d_dataB, B, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
	dim3 threadPerBlock(20, 20);
	dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y);
	//printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
	matrix_mul_gpu <<< blockNumber, threadPerBlock >>> (d_dataA, d_dataB, d_dataC, Col);
	//������������-һ������ָ��
	hipMemcpy(C, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);
	cout << d_dataC << endl;
	//�ͷ��ڴ�
	free(A);
	free(B);
	free(C);
	hipFree(d_dataA);
	hipFree(d_dataB);
	hipFree(d_dataC);

	endTime = clock();//��ʱ����

	cout << "GPU The run time is: " << (double)(endTime - startTime) /* CLOCKS_PER_SEC*/ << "ms" << endl;
	return 0;
}
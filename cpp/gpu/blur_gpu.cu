#include "hip/hip_runtime.h"
#include "blur_gpu.cuh"

__global__ void operatepic(int g, int *img1, int *img2, int *img3, int index) {
	int t = 32 * 30;
	int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
	int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
	int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockId_3D;

	if (g / t < 1) {
		img3[i] = (img1[i] + img2[i]) / 2;
	}
	else {
		for (int j = 0; j <= (g / t); j++) {
			if (index == 0) {
				img3[i + j * t] = (img1[i + j * t] + img2[i + j * t]) / 2;
			}
			else
			{
				img3[i + j * t] = (img1[i + j * t] + img2[i + j * t]) / 2 + 1;
			}
		}
	}
}


int main(int index = 0) {
	clock_t startTime, endTime;
	startTime = clock();
	int Row = 1920;
	int Col = 1080;
	int *A = (int *)malloc(sizeof(int) * Row * Col);
	int *B = (int *)malloc(sizeof(int) * Row * Col);
	int *C = (int *)malloc(sizeof(int) * Row * Col);
	//malloc device memory
	int *d_dataA, *d_dataB, *d_dataC;
	//hipSetDevice(3);
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(index);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
	}
	else {
		printf("success\n");
	}
	hipMalloc((void**)&d_dataA, sizeof(int) * Row * Col);
	hipMalloc((void**)&d_dataB, sizeof(int) * Row * Col);
	hipMalloc((void**)&d_dataC, sizeof(int) * Row * Col);
	//set value
	for (int i = 0; i < Row * Col; i++) {
		A[i] = 255;
		B[i] = 0;
	}
	int s = Row * Col;
	//cout << s << endl;
	hipMemcpy(d_dataA, A, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
	hipMemcpy(d_dataB, B, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
	dim3 threadPerBlock(32, 1, 1);
	dim3 blockNumber(30, 1, 1);
	//printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);

	operatepic <<< blockNumber, threadPerBlock >>> (s, d_dataA, d_dataB, d_dataC, index);
    //������������-һ������ָ��
	hipMemcpy(C, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);
//	for (int j = 0; j < s; j++) {
//		printf("%d", C[j]);
//	}
    //�ͷ��ڴ�
	free(A);
	free(B);
	free(C);
	hipFree(d_dataA);
	hipFree(d_dataB);
	hipFree(d_dataC);

	endTime = clock();//��ʱ����

	cout << "GPU The run time is: " << (double)(endTime - startTime) /* CLOCKS_PER_SEC*/ << "ms" << endl;
	return 0;
}